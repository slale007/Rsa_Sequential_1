#include "hip/hip_runtime.h"
/////////////////////////////////////
// My implementation of Montgomery //
/////////////////////////////////////

#include "..\Rsa_Sequential_1\stdafx.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <string.h>

#include "..\Rsa_Cuda_1\rsa1.h"
#include "..\Rsa_Sequential_1\timeutil.h"
#include "..\Rsa_Cuda_1\customFunctions.h"
#include "cudaMpir.h"
#include "hip/hip_runtime.h"
#include ""
#define BASE 64

using namespace std;

clock_t globalTime0;
clock_t globalTime1;
clock_t globalTime2;
clock_t globalTime3;
clock_t globalTime4;
clock_t globalTime5;
clock_t globalTime6;
clock_t globalTime7;
clock_t globalTime8;
clock_t start;

// This one is correct!
void MontgomeryModularMultiplicationV4(mpz_t res, mpz_t xxx, mpz_t yyy, mpz_t modul, mpz_t mprim, mpz_t R, int index)
{
	mpz_t t;
	mpz_init(t);
	mpz_t t2;
	mpz_init(t2);
	mpz_t tmp1;
	mpz_init(tmp1);
	mpz_t tmp2;
	mpz_init(tmp2);
	mpz_t tmp3;
	mpz_init(tmp3);
	mpz_t tmp4;
	mpz_init(tmp4);
	mpz_t u;
	mpz_init(u);
	mpz_t u2;
	mpz_init(u2);
	mpz_t slowU;
	mpz_init(slowU);

    start = clock();

	mpz_mul(t, xxx, yyy);
	//Multiplication(t2, xxx, yyy);

	if (mpz_cmp(t, t2) != 0) {
		/*cout << std::hex << "Mul--- Fatal Error" << endl;
		/*cout << "Good value: "<<endl;
		for (int i = 0; i < t->_mp_size; i++) {
			cout << t->_mp_d[i] << endl;
		}
		cout <<  "Bad value: " << endl;
		for (int i = 0; i < t2->_mp_size; i++) {
			cout << t2->_mp_d[i] << endl;
		}*/
	}

	globalTime0 += clock() - start;
	start = clock();

	mpz_mul(tmp1, t, mprim);

	globalTime1 += clock() - start;
	start = clock();

	// mpz_mod(tmp2, tmp1, R);
	mpz_tdiv_q_2exp(tmp2, tmp1, index);
	mpz_mul_2exp(tmp2, tmp2, index);
	mpz_sub(tmp2, tmp1, tmp2);

	globalTime4 += clock() - start;
	start = clock();

	mpz_mul(tmp3, tmp2, modul);

	globalTime5 += clock() - start;
	start = clock();

	mpz_add(tmp4, t, tmp3);

	globalTime6 += clock() - start;
	start = clock();

    mpz_tdiv_q_2exp(u, tmp4, index);
	// RightShiftBlocks(u, tmp4, index/64);

	if (mpz_cmp(u, u2) != 0) {
		// cout << "Mod--- Fatal Error" << endl;
	}
	else {
		// cout << "Mod--- All Right" << endl;
	}

	globalTime7 += clock() - start;
	start = clock();

	// step 3.
	if (mpz_cmp(u, modul) >= 0)
	{
		mpz_sub(res, u, modul);
	}
	else {
		mpz_add_ui(res, u, 0); // ok
	}

	globalTime8 += clock() - start;
}

void MontgomeryModularEponentiationV4(mpz_t res, mpz_t xxx, mpz_t exponent, mpz_t modul)
{
	mpz_t tempNull;
	mpz_init(tempNull);
	mpz_add_ui(tempNull, tempNull, 0);

	// Calculate R and xline = Rmod = R mod modul
	mpz_t RR;
	mpz_t RMod;
	mpz_t Rsquare;
	mpz_t RsquareMod;
	mpz_init(RR);
	mpz_init(RMod);
	mpz_init(Rsquare);
	mpz_init(RsquareMod);
	mpz_add_ui(RR, RR, 1);
	int indexpom0 = 0;

	mpz_mul_2exp(RR, RR, modul->_mp_size * 64);
	mpz_mul_2exp(Rsquare, RR, 1);
	mpz_mod(RsquareMod, Rsquare, modul);
	mpz_mod(RMod, RR, modul);

	// above is correct

	mpz_t mprim;
	mpz_init(mprim);
	mpz_t mprim2;
	mpz_init(mprim2);
	mpz_t base;
	mpz_init(base);
	mpz_set_ui(base, 2);

	mpz_t min1;
	mpz_t onne;
	mpz_init(onne);
	mpz_add_ui(onne, onne, 1);
	mpz_init(min1);
	mpz_sub(min1, tempNull, onne);
	mpz_powm(mprim, modul, min1, RR);

	mpz_sub(mprim, RR, mprim);


	mpz_t xline;
	mpz_init(xline);
	mpz_t xline2pom;
	mpz_init(xline2pom);
	mpz_t xline2;
	mpz_init(xline2);
	mpz_mul(xline2pom, xxx, RR);
	mpz_mod(xline, xline2pom, modul);

	// MontgomeryModularMultiplicationV4(xline, xxx, RsquareMod, modul, mprim, RR);

	if (mpz_cmp(xline, xline2) != 0) {
		// cout << endl << "Fatal error02" << endl;
	}

	mpz_mod(res, RR, modul);

	int indexpom = 0;
	for (int i = 63; i >= 0; i--) {
		if (exponent->_mp_d[exponent->_mp_size - 1] & (((unsigned long long int)1) << i)) {
			indexpom = i;
			break;
		}
	}
	int index = 64 * exponent->_mp_size - 64 + indexpom; // ok

	int indexRR = 64 * RR->_mp_size - 64;

	for (int i = index; i >= 0; i--) {
		MontgomeryModularMultiplicationV4(res, res, res, modul, mprim, RR, indexRR);
		if (exponent->_mp_d[i / 64] & (((unsigned long long int)1) << (i % 64))) {
			MontgomeryModularMultiplicationV4(res, res, xline, modul, mprim, RR, indexRR);
		}
	}

	cout << "Global time 0: " << globalTime0 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 1: " << globalTime1 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 2: " << globalTime2 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 3: " << globalTime3 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 4: " << globalTime4 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 5: " << globalTime5 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 6: " << globalTime6 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 7: " << globalTime7 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 8: " << globalTime8 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;

	// all above stuff is checked
	mpz_t one;
	mpz_t AAA;
	mpz_init(one);
	mpz_init(AAA);
	mpz_add(AAA, tempNull, res);

	mpz_add_ui(one, one, 1);
	MontgomeryModularMultiplicationV4(res, AAA, one, modul, mprim, RR, indexRR);
}

void rsac_encryptV4(public_key *pub, const char *message, size_t m_len, char **cryptedMessage, size_t *c_len)
{
	mpz_t m_int, c_int, c_int2, c_int3;
	mpz_inits(m_int, c_int, c_int2, c_int3, NULL);
	mpz_import(m_int, m_len, /* MS word first */ 1, /* bytes per word */ 1, /* big-endian */ 1, /* skip bits */ 0, message);

	clock_t startTime = std::clock();
	MontgomeryModularEponentiationV4(/*cripted*/c_int,/* message */ m_int, /*exponent*/ pub->e, /*modul*/ pub->n);
	cout << "Montgomery realization: ";
	printTime(startTime);


	startTime = std::clock();
	rsac_encrypt_internal(pub, m_int, c_int2);
	cout << "Mpir realization: ";
	printTime(startTime);



	*cryptedMessage = (char*)mpz_export(NULL, c_len, 1, 1, 1, 0, c_int);
	// mpz_clears(m_int, c_int, NULL);
}

void rsac_decryptV4(private_key *priv, const char *c, size_t c_len, char **m, size_t *m_len)
{
	mpz_t m_int, c_int, m_int2;
	mpz_inits(m_int, c_int, m_int2, NULL);
	mpz_import(
		c_int, c_len, /* MS word first */ 1, /* bytes per word */ 1,
		/* big-endian */ 1, /* skip bits */ 0, c);


	clock_t startTime = std::clock();
	MontgomeryModularEponentiationV4(/*cripted*/m_int,/* message */ c_int, /*exponent*/ priv->d, /*modul*/ priv->n);
	cout << "Montgomery realization: ";
	printTime(startTime);

	// Mpir realization of powm
	startTime = std::clock();
	rsac_decrypt_internal(priv, c_int, m_int);
	cout << "Mpir realization: ";
	printTime(startTime);


	*m = (char*)mpz_export(NULL, m_len, 1, 1, 1, 0, m_int);
	mpz_clears(m_int, c_int, m_int2, NULL);
}

int test_rsac_string_encrypt_decrypt4() {
	char message[] = "Stop slacking off.Stop slacking off.Stop slacking off.Stop slacking off.";
	size_t c_len, m_len = strlen(message), result_len;
	char **c = (char**)calloc(sizeof(char *), 1);
	char **m_result = (char**)calloc(sizeof(char *), 1);
	int fail = 0;
	public_key* pub = (public_key*)calloc(sizeof(public_key), 1);
	private_key* priv = (private_key*)calloc(sizeof(private_key), 1);

	if (pub == NULL || priv == NULL) {
		printf("FAIL: rsac_string_encrypt_decrypt could not allocate public or private key struct\n");
		return 1;
	}

	printf("\n_______________________Key generation_______________________\n\n");

	// Initialize public key
	mpz_init(pub->n);
	mpz_init(pub->e);
	// Initialize private key
	mpz_init(priv->n);
	mpz_init(priv->e);
	mpz_init(priv->d);
	mpz_init(priv->p);
	mpz_init(priv->q);

	clock_t keygenTime = std::clock();
	int res = rsac_keygen(pub, priv);
	// int res = generate_keys(pub, priv);

	if (res != 0) {
		printf("FAIL: rsac_string_encrypt_decrypt rsac_keygen returned %d, expected 0\n", res);
		fail++;
	}
	printTime(keygenTime);

	printf("\n_________________________Encription_________________________\n\n");

	rsac_encryptV4(pub, message, m_len, c, &c_len);

	printf("\n_________________________Decription_________________________\n\n");

	rsac_decryptV4(priv, *c, c_len, m_result, &result_len);

	printf("\n________________________Final Result________________________\n\n");
	printf("expected:\n'%s' \ngot:\n'%s'\n", message, *m_result);

	free(pub);
	free(priv);
	free(*c);
	free(*m_result);
	if (fail == 0) {
		printf("\nTest PASSED\n");
	}

	return fail;
}

void printGPUProperties() {
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		cout << "  Number of multiprocesors: " << prop.multiProcessorCount << endl;
		cout << "  Total global memory: " << prop.totalGlobalMem << endl;
		cout << "  Shared memory per block: " << prop.sharedMemPerBlock << endl;
		cout << "  Number of registers per block per block: " << prop.regsPerBlock << endl;
		cout << "  Warp size: " << prop.warpSize << endl;
		cout << "  Max threads per block: " << prop.maxThreadsPerBlock << endl;
	}
	cout << endl;
}

int main() {
	int failures = 0;
	printGPUProperties();
	printf(" CHAR_BIT je: %d\n", CHAR_BIT);
	printf("Velicina char je: %d\n", sizeof(char));
	printf("Velicina unsigned char je: %d\n", sizeof(unsigned char));
	printf("Velicina mp_limb_t je: %d\n", sizeof(mp_limb_t));
	printf("Velicina unsigned long int je: %d\n", sizeof(unsigned long int));
	printf("Velicina unsigned long long int je: %d\n", sizeof(unsigned long long int));

	failures += test_rsac_string_encrypt_decrypt4();

	printf("%d failures\n", failures);
	return failures > 0;
}