#include "hip/hip_runtime.h"
﻿/////////////////////////////////////
// My implementation of Montgomery //
/////////////////////////////////////

#include "..\Rsa_Sequential_1\stdafx.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <string.h>

#include "..\Rsa_Cuda_1\rsa1.h"
#include "..\Rsa_Sequential_1\timeutil.h"
#include "..\Rsa_Cuda_1\customFunctions.h"
#include "cudaMpir.h"
#include "hip/hip_runtime.h"
#include ""
#define BASE 64

using namespace std;

clock_t globalTime0;
clock_t globalTime1;
clock_t globalTime2;
clock_t globalTime3;
clock_t globalTime4;
clock_t globalTime5;
clock_t globalTime6;
clock_t globalTime7;
clock_t globalTime8;
clock_t start;

// Note: need to test with big strings
char oldMessageForTesting[] = "Stop slacking off.Stop slacking off.Stop slacking off.Stop slacking off.";
char messageForTesting[] = "Nikola Tesla je umro. Umro je siromasan, ali je bio jedan od najkorisnijih ljudi koji su ikada ziveli. Ono sto je stvorio veliko je i, kako vreme prolazi, postaje jos vece";

// This one is correct!
void MontgomeryModularMultiplicationV4(mpz_t res, mpz_t xxx, mpz_t yyy, mpz_t modul, mpz_t mprim, mpz_t R, int index)
{
	mpz_t t;
	mpz_init(t);
	mpz_t t2;
	mpz_init(t2);
	mpz_t tmp1;
	mpz_init(tmp1);
	mpz_t tmp2;
	mpz_init(tmp2);
	mpz_t tmp3;
	mpz_init(tmp3);
	mpz_t tmp4;
	mpz_init(tmp4);
	mpz_t u;
	mpz_init(u);
	mpz_t u2;
	mpz_init(u2);
	mpz_t slowU;
	mpz_init(slowU);

    start = clock();

	mpz_mul(t, xxx, yyy);
	//Multiplication(t2, xxx, yyy);

	if (mpz_cmp(t, t2) != 0) {
		/*cout << std::hex << "Mul--- Fatal Error" << endl;
		/*cout << "Good value: "<<endl;
		for (int i = 0; i < t->_mp_size; i++) {
			cout << t->_mp_d[i] << endl;
		}
		cout <<  "Bad value: " << endl;
		for (int i = 0; i < t2->_mp_size; i++) {
			cout << t2->_mp_d[i] << endl;
		}*/
	}

	globalTime0 += clock() - start;
	start = clock();

	mpz_mul(tmp1, t, mprim);

	globalTime1 += clock() - start;
	start = clock();

	// mpz_mod(tmp2, tmp1, R);
	mpz_tdiv_q_2exp(tmp2, tmp1, index);
	mpz_mul_2exp(tmp2, tmp2, index);
	mpz_sub(tmp2, tmp1, tmp2);

	globalTime4 += clock() - start;
	start = clock();

	mpz_mul(tmp3, tmp2, modul);

	globalTime5 += clock() - start;
	start = clock();

	mpz_add(tmp4, t, tmp3);

	globalTime6 += clock() - start;
	start = clock();

    mpz_tdiv_q_2exp(u, tmp4, index);
	// RightShiftBlocks(u, tmp4, index/64);

	if (mpz_cmp(u, u2) != 0) {
		// cout << "Mod--- Fatal Error" << endl;
	}
	else {
		// cout << "Mod--- All Right" << endl;
	}

	globalTime7 += clock() - start;
	start = clock();

	// step 3.
	if (mpz_cmp(u, modul) >= 0)
	{
		mpz_sub(res, u, modul);
	}
	else {
		mpz_add_ui(res, u, 0); // ok
	}

	globalTime8 += clock() - start;
}

void MontgomeryModularExponentiationV4(mpz_t res, mpz_t xxx, mpz_t exponent, mpz_t modul)
{
	mpz_t tempNull;
	mpz_init(tempNull);
	mpz_add_ui(tempNull, tempNull, 0);

	// Calculate R and xline = Rmod = R mod modul
	mpz_t RR;
	mpz_t RMod;
	mpz_t Rsquare;
	mpz_t RsquareMod;
	mpz_init(RR);
	mpz_init(RMod);
	mpz_init(Rsquare);
	mpz_init(RsquareMod);
	mpz_add_ui(RR, RR, 1);
	int indexpom0 = 0;

	// Calculate R: R = b ^ messageLength
	mpz_mul_2exp(RR, RR, modul->_mp_size * 64);

	mpz_mul_2exp(Rsquare, RR, 1);
	mpz_mod(RsquareMod, Rsquare, modul);
	mpz_mod(RMod, RR, modul);

	mpz_t mprim;
	mpz_init(mprim);

	mpz_t min1;
	mpz_t onne;
	mpz_init(onne);
	mpz_add_ui(onne, onne, 1);
	mpz_init(min1);
	mpz_sub(min1, tempNull, onne);
	mpz_powm(mprim, modul, min1, RR);

	mpz_sub(mprim, RR, mprim);


	mpz_t xline;
	mpz_init(xline);
	mpz_t xline2pom;
	mpz_init(xline2pom);
	mpz_t xline2;
	mpz_init(xline2);
	mpz_mul(xline2pom, xxx, RR);
	mpz_mod(xline, xline2pom, modul);


	int indexpom = 0;
	for (int i = 63; i >= 0; i--) {
		if (exponent->_mp_d[exponent->_mp_size - 1] & (((unsigned long long int)1) << i)) {
			indexpom = i;
			break;
		}
	}
	int index = 64 * exponent->_mp_size - 64 + indexpom; // ok

	int indexRR = 64 * RR->_mp_size - 64;



	MontgomeryModularMultiplicationV4(xline2, xxx, RsquareMod, modul, mprim, RR, indexRR);

	if (mpz_cmp(xline, xline2) != 0) {
		cout << endl << "Not same" << endl;
	}

	mpz_mod(res, RR, modul);





	for (int i = index; i >= 0; i--) {
		MontgomeryModularMultiplicationV4(res, res, res, modul, mprim, RR, indexRR);
		if (exponent->_mp_d[i / 64] & (((unsigned long long int)1) << (i % 64))) {
			MontgomeryModularMultiplicationV4(res, res, xline, modul, mprim, RR, indexRR);
		}
	}

	cout << "Global time 0: " << globalTime0 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 1: " << globalTime1 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 2: " << globalTime2 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 3: " << globalTime3 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 4: " << globalTime4 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 5: " << globalTime5 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 6: " << globalTime6 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 7: " << globalTime7 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;
	cout << "Global time 8: " << globalTime8 / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;

	// all above stuff is checked
	mpz_t one;
	mpz_t AAA;
	mpz_init(one);
	mpz_init(AAA);
	mpz_add(AAA, tempNull, res);

	mpz_add_ui(one, one, 1);
	MontgomeryModularMultiplicationV4(res, AAA, one, modul, mprim, RR, indexRR);
}

void rsaEncryption(public_key *publicKey, const char *message, size_t messageLength, char **cryptedMessage, size_t *ciphertextLength)
{
	mpz_t originalMessage, ciphertext, ciphertext2, c_int3;
	mpz_inits(originalMessage, ciphertext, ciphertext2, c_int3, NULL);
	mpz_import(originalMessage,
		messageLength, 
		/* MS word first */ 1,
		/* bytes per word */ 1,
		/* big-endian */ 1,
		/* skip bits */ 0,
		message);

	clock_t startTime = std::clock();
	MontgomeryModularExponentiationV4(
		/* cripted*/ciphertext,
		/* message */ originalMessage,
		/* exponent*/ publicKey->e,
		/* modul*/ publicKey->n);
	cout << "Montgomery realization: "; printTime(startTime);

	startTime = std::clock();
	rsac_encrypt_internal(publicKey, originalMessage, ciphertext2);
	cout << "Mpir realization: "; printTime(startTime);


	*cryptedMessage = (char*)mpz_export(NULL, ciphertextLength, 1, 1, 1, 0, ciphertext);
}

void rsaDecryption(private_key *priv, const char *c, size_t c_len, char **m, size_t *m_len)
{
	mpz_t m_int, c_int, m_int2;
	mpz_inits(m_int, c_int, m_int2, NULL);
	mpz_import(
		c_int, c_len, /* MS word first */ 1, /* bytes per word */ 1,
		/* big-endian */ 1, /* skip bits */ 0, c);


	clock_t startTime = std::clock();
	MontgomeryModularExponentiationV4(/*cripted*/m_int,/* message */ c_int, /*exponent*/ priv->d, /*modul*/ priv->n);
	cout << "Montgomery realization: ";
	printTime(startTime);

	// Mpir realization of powm
	startTime = std::clock();
	rsac_decrypt_internal(priv, c_int, m_int);
	cout << "Mpir realization: ";
	printTime(startTime);


	*m = (char*)mpz_export(NULL, m_len, 1, 1, 1, 0, m_int);
	mpz_clears(m_int, c_int, m_int2, NULL);
}

void testRsaSequentialMontgomery() {
	char* message = messageForTesting;
	size_t ciphertextLength, messageLength = strlen(message), result_len;
	char **c = (char**)calloc(sizeof(char *), 1);
	char **m_result = (char**)calloc(sizeof(char *), 1);
	int fail = 0;
	public_key* publicKey = (public_key*)calloc(sizeof(public_key), 1);
	private_key* privateKey = (private_key*)calloc(sizeof(private_key), 1);

	printf("\n_______________________Key generation_______________________\n\n");

	// Initialize public and private key
	mpz_init(publicKey->n);
	mpz_init(publicKey->e);
	mpz_init(privateKey->n);
	mpz_init(privateKey->e);
	mpz_init(privateKey->d);
	mpz_init(privateKey->p);
	mpz_init(privateKey->q);

	clock_t keygenTime = std::clock();
	rsaKeyGeneration(publicKey, privateKey);
	printTime(keygenTime);

	printf("\n_________________________Encription_________________________\n\n");

	rsaEncryption(publicKey, message, messageLength, c, &ciphertextLength);

	printf("\n_________________________Decription_________________________\n\n");

	rsaDecryption(privateKey, *c, ciphertextLength, m_result, &result_len);

	printf("\n________________________Final Result________________________\n\n");
	printf("expected:\n'%s' \ngot:\n'%s'\n", message, *m_result);

	free(publicKey);
	free(privateKey);
	free(*c);
	free(*m_result);

	if (fail == 0) {
		printf("\nTest PASSED\n");
	}
}

void printGPUProperties() {
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		cout << "  Number of multiprocesors: " << prop.multiProcessorCount << endl;
		cout << "  Total global memory: " << prop.totalGlobalMem << endl;
		cout << "  Shared memory per block: " << prop.sharedMemPerBlock << endl;
		cout << "  Number of registers per block per block: " << prop.regsPerBlock << endl;
		cout << "  Warp size: " << prop.warpSize << endl;
		cout << "  Max threads per block: " << prop.maxThreadsPerBlock << endl;
	}
	cout << endl;
}

void printSomeDebuggingStuff() {
	printf("CHAR_BIT je: %d\n", CHAR_BIT);
	printf("Velicina char je: %d\n", sizeof(char));
	printf("Velicina unsigned char je: %d\n", sizeof(unsigned char));
	printf("Velicina unsigned short je: %d\n", sizeof(unsigned short));
	printf("Velicina unsigned int je: %d\n", sizeof(unsigned int));
	printf("Velicina mp_limb_t je: %d\n", sizeof(mp_limb_t));
	printf("Velicina size_t je: %d\n", sizeof(size_t));
	printf("Velicina unsigned long int je: %d\n", sizeof(unsigned long int));
	printf("Velicina unsigned long long je: %d\n", sizeof(unsigned long long));
	printf("Velicina unsigned long long int je: %d\n", sizeof(unsigned long long int));
}

int main() {

	printGPUProperties();
	printSomeDebuggingStuff();

	testRsaSequentialMontgomery();

	return 0;
}