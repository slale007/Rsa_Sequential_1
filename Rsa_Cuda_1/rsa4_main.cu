#include "hip/hip_runtime.h"
﻿/////////////////////////////////////
// My implementation of Montgomery //
/////////////////////////////////////

#include "..\Rsa_Sequential_1\stdafx.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <string.h>

#include "..\Rsa_Cuda_1\rsa1.h"
#include "..\Rsa_Sequential_1\timeutil.h"
#include "..\Rsa_Cuda_1\customFunctions.h"
#include "cudaMpir.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include ""
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

#define LONGINT int

using namespace std;

clock_t globalTime0;
clock_t globalTime1;
clock_t globalTime2;
clock_t globalTime3;
clock_t globalTime4;
clock_t globalTime5;
clock_t globalTime6;
clock_t globalTime7;
clock_t globalTime8;
clock_t globalTime9;
clock_t start;

// Note: need to test with big strings
char oldMessageForTesting[] = "Stop slacking off.Stop slacking off.Stop slacking off.Stop slacking off.";
char messageForTesting[] = "Nikola Tesla je umro. Umro je siromasan, ali je bio jedan od najkorisnijih ljudi koji su ikada ziveli. Ono sto je stvorio veliko je i, kako vreme prolazi, postaje jos vece";





__global__ void cuda_Multiplication(LONGINT* result, unsigned char* first, unsigned char* second, int lengthFirst, int lengthSecond) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ unsigned char shared_First[2048];
	__shared__ unsigned char shared_Second[2048];

	for (int i = threadIdx.x; i < lengthFirst; i += blockDim.x) {
		shared_First[i] = first[i];
	}

	for (int i = threadIdx.x; i < lengthSecond; i += blockDim.x) {
		shared_Second[i] = second[i];
	}

	__syncthreads();

	if (idx < lengthFirst) {
		int m = 0;
		int n = idx;
		int tmp = 0;

		while (n >= 0 && m < lengthSecond) {
			tmp += shared_Second[m] * shared_First[n];
			m++;
			n--;
		}

		result[idx] = tmp;
	}
	else if (idx < lengthFirst + lengthSecond - 1) {
		int n = lengthFirst - 1;
		int m = idx - n;
		int tmp = 0;

		while (m < lengthSecond && n >= 0) {
			tmp += shared_Second[m] * shared_First[n];
			m++;
			n--;
		}

		result[idx] = tmp;
	}
}

__global__ void cuda_CarryUpdate(LONGINT* longResult, int* lengthLongResult) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx == 0) {
		int len = *lengthLongResult;
		int carry = 0;
		int tmp = 0;
		int i;
		for (i = 0; i < len; i++) {
			tmp = longResult[i] + carry;
			carry = tmp >> 8;
			longResult[i] = tmp & 0xff;
		}

		if (carry != 0) {
			longResult[i] = carry;
			*lengthLongResult = i + 1;
		}
	}
}

// Is this tested?   
//
////
void MultiplicationInCuda(mpz_t result, mpz_t first, mpz_t second) {
	hipError_t cudaStatus;
	int length1 = first->_mp_size * 8;
	int length2 = second->_mp_size * 8;

	mpz_init(result);
	result->_mp_size = (length1 + length2);
	result->_mp_alloc = result->_mp_size;
	result->_mp_d = (unsigned long long int *)malloc(result->_mp_size * sizeof(unsigned long long int));

	int *tmpRes = (int *)malloc((length1 + length2) * sizeof(int));

	unsigned char* dev_first;
	unsigned char* dev_second;
	LONGINT* dev_result;

	cudaStatus = hipMalloc((void**)&dev_result, (length1 + length2) * sizeof(LONGINT));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed-1!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_first, length1 * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed0!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_second, length2 * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed1!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_first, first->_mp_d, length1 * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy first failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_second, second->_mp_d, length2 * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy second failed!");
		goto Error;
	}


	int numberOfThreads = length1 + length2;

	// Launch a kernel on the GPU with one thread for each element.
	cuda_Multiplication <<< 1, numberOfThreads >>>(dev_result, dev_first, dev_second, length1, length2);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Multiplication launch failed123: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "2cudaDeviceSynchronize returned error code %d after launching cuda_RightShiftsBlocks!\n", cudaStatus);
		goto Error;
	}

	// carry update on cuda

	/*int *midLength;
	int realMidLength = length1 + length2;


	cudaStatus = hipMalloc((void**)&midLength, sizeof(int));
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed0!");
	goto Error;
	}

	cudaStatus = hipMemcpy(midLength, &realMidLength, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy second failed!");
	goto Error;
	}

	cuda_CarryUpdate <<<1, 1 >>> (dev_result, midLength);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "Multiplication launch failed1234: %s\n", hipGetErrorString(cudaStatus));
	goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "23cudaDeviceSynchronize returned error code %d after launching cuda_RightShiftsBlocks!\n", cudaStatus);
	goto Error;
	}*/

	// END carry update on cuda 




	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(tmpRes, dev_result, (length1 + length2) * sizeof(LONGINT), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "2cudaMemcpy failed!\n");
		goto Error;
	}

	int len = length1 + length2 - 1;
	unsigned long long int carry = 0;
	unsigned long long int tmp = 0;
	int i;
	for (i = 0; i < len; i++) {
		tmp = tmpRes[i] + carry;
		carry = tmp >> 8;
		tmpRes[i] = tmp & 0xff;
	}

	if (carry != 0) {
		tmpRes[i] = carry;
		len++;
	}

	result->_mp_d[0] = tmpRes[0];
	result->_mp_d[0] |= tmpRes[1] << 8;
	result->_mp_d[0] |= (unsigned long long int)tmpRes[2] << 2 * 8;
	result->_mp_d[0] |= (unsigned long long int)tmpRes[3] << 3 * 8;
	result->_mp_d[0] |= (unsigned long long int)tmpRes[4] << 4 * 8;
	result->_mp_d[0] |= (unsigned long long int)tmpRes[5] << 5 * 8;
	result->_mp_d[0] |= (unsigned long long int)tmpRes[6] << 6 * 8;
	result->_mp_d[0] |= (unsigned long long int)tmpRes[7] << 7 * 8;

	for (int k = 1; k < (len+1) / 8; k++) {
		result->_mp_d[k] = 0;
		result->_mp_d[k] |= tmpRes[8 * k];
		result->_mp_d[k] |= tmpRes[8 * k + 1] << 8;
		result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 2] << 2 * 8;
		result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 3] << 3 * 8;
		result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 4] << 4 * 8;
		result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 5] << 5 * 8;
		result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 6] << 6 * 8;
		result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 7] << 7 * 8;
	}

	result->_mp_size = (length1 + length2) / 8;
	result->_mp_alloc = (length1 + length2) / 8;

Error:
	hipFree(dev_first);
	hipFree(dev_second);
	hipFree(dev_result);
}






























































// This one is correct!
void MontgomeryModularMultiplicationV4(mpz_t res, mpz_t xxx, mpz_t yyy, mpz_t modul, mpz_t mprim, mpz_t R, int index)
{
	mpz_t t;
	mpz_init(t);
	mpz_t t2;
	mpz_init(t2);
	mpz_t tmp1;
	mpz_init(tmp1);
	mpz_t tmp2;
	mpz_init(tmp2);
	mpz_t tmp3;
	mpz_init(tmp3);
	mpz_t tmp4;
	mpz_init(tmp4);
	mpz_t u;
	mpz_init(u);
	mpz_t slowU;
	mpz_init(slowU);

	// START tmp data
	/*xxx->_mp_d[0] = 256;
	xxx->_mp_size = 1;

	yyy->_mp_d[0] = 256;
	yyy->_mp_size = 1;
	*/
	// END tmp data

	//mpz_mul(t, xxx, yyy);

	MultiplicationInCuda(t, xxx, yyy);

	/*if (mpz_cmp(t, t2) != 0) {
		cout << endl << "Nooooooooooooooooooooooooooooot same" << endl;
	}*/

	mpz_mul(tmp1, t, mprim);

	mpz_tdiv_q_2exp(tmp2, tmp1, index);
	mpz_mul_2exp(tmp2, tmp2, index);
	mpz_sub(tmp2, tmp1, tmp2);

	mpz_mul(tmp3, tmp2, modul);

	mpz_add(tmp4, t, tmp3);

    mpz_tdiv_q_2exp(u, tmp4, index);

	// step 3.
	if (mpz_cmp(u, modul) >= 0)
	{
		mpz_sub(res, u, modul);
	}
	else {
		mpz_add_ui(res, u, 0); // ok
	}
}

void MontgomeryModularExponentiationV4(mpz_t res, mpz_t xxx, mpz_t exponent, mpz_t modul)
{
	mpz_t tempNull;
	mpz_init(tempNull);
	mpz_add_ui(tempNull, tempNull, 0);

	// Calculate R and xline = Rmod = R mod modul
	mpz_t RR;
	mpz_t RMod;
	mpz_t Rsquare;
	mpz_t RsquareMod;
	mpz_init(RR);
	mpz_init(RMod);
	mpz_init(Rsquare);
	mpz_init(RsquareMod);
	mpz_add_ui(RR, RR, 1);
	int indexpom0 = 0;

	// Calculate R: R = b ^ messageLength
	mpz_mul_2exp(RR, RR, modul->_mp_size * 64);

	mpz_mul_2exp(Rsquare, RR, 1);
	mpz_mod(RsquareMod, Rsquare, modul);
	mpz_mod(RMod, RR, modul);

	mpz_t mprim;
	mpz_init(mprim);

	mpz_t min1;
	mpz_t onne;
	mpz_init(onne);
	mpz_add_ui(onne, onne, 1);
	mpz_init(min1);
	mpz_sub(min1, tempNull, onne);
	mpz_powm(mprim, modul, min1, RR);

	mpz_sub(mprim, RR, mprim);


	mpz_t xline;
	mpz_init(xline);
	mpz_t xline2pom;
	mpz_init(xline2pom);
	mpz_t xline2;
	mpz_init(xline2);
	mpz_mul(xline2pom, xxx, RR);
	mpz_mod(xline, xline2pom, modul);


	int indexpom = 0;
	for (int i = 63; i >= 0; i--) {
		if (exponent->_mp_d[exponent->_mp_size - 1] & (((unsigned long long int)1) << i)) {
			indexpom = i;
			break;
		}
	}
	int index = 64 * exponent->_mp_size - 64 + indexpom; // ok

	int indexRR = 64 * RR->_mp_size - 64;



	/*MontgomeryModularMultiplicationV4(xline2, xxx, RsquareMod, modul, mprim, RR, indexRR);

	if (mpz_cmp(xline, xline2) != 0) {
		cout << endl << "Nooooooooooooooooooooooooooooot same" << endl;
	}*/

	mpz_mod(res, RR, modul);

	for (int i = index; i >= 0; i--) {
		MontgomeryModularMultiplicationV4(res, res, res, modul, mprim, RR, indexRR);
		if (exponent->_mp_d[i / 64] & (((unsigned long long int)1) << (i % 64))) {
			MontgomeryModularMultiplicationV4(res, res, xline, modul, mprim, RR, indexRR);
		}
	}

	// all above stuff is checked
	mpz_t one;
	mpz_t AAA;
	mpz_init(one);
	mpz_init(AAA);
	mpz_add(AAA, tempNull, res);

	mpz_add_ui(one, one, 1);
	MontgomeryModularMultiplicationV4(res, AAA, one, modul, mprim, RR, indexRR);
}

void rsaEncryption(public_key *publicKey, const char *message, size_t messageLength, char **cryptedMessage, size_t *ciphertextLength)
{
	mpz_t originalMessage, ciphertext, ciphertext2, c_int3;
	mpz_inits(originalMessage, ciphertext, ciphertext2, c_int3, NULL);
	mpz_import(originalMessage,
		messageLength, 
		/* MS word first */ 1,
		/* bytes per word */ 1,
		/* big-endian */ 1,
		/* skip bits */ 0,
		message);

	clock_t startTime = std::clock();
	MontgomeryModularExponentiationV4(
		/* cripted*/ciphertext,
		/* message */ originalMessage,
		/* exponent*/ publicKey->e,
		/* modul*/ publicKey->n);
	cout << "Montgomery realization: "; printTime(startTime);

	startTime = std::clock();
	rsac_encrypt_internal(publicKey, originalMessage, ciphertext2);
	cout << "Mpir realization: "; printTime(startTime);


	*cryptedMessage = (char*)mpz_export(NULL, ciphertextLength, 1, 1, 1, 0, ciphertext);
}

void rsaDecryption(private_key *priv, const char *c, size_t c_len, char **m, size_t *m_len)
{
	mpz_t m_int, c_int, m_int2;
	mpz_inits(m_int, c_int, m_int2, NULL);
	mpz_import(
		c_int, c_len, /* MS word first */ 1, /* bytes per word */ 1,
		/* big-endian */ 1, /* skip bits */ 0, c);


	clock_t startTime = std::clock();
	MontgomeryModularExponentiationV4(/*cripted*/m_int,/* message */ c_int, /*exponent*/ priv->d, /*modul*/ priv->n);
	cout << "Montgomery realization: ";
	printTime(startTime);

	// Mpir realization of powm
	startTime = std::clock();
	rsac_decrypt_internal(priv, c_int, m_int);
	cout << "Mpir realization: ";
	printTime(startTime);


	*m = (char*)mpz_export(NULL, m_len, 1, 1, 1, 0, m_int);
	mpz_clears(m_int, c_int, m_int2, NULL);
}

void testRsaSequentialMontgomery() {
	char* message = messageForTesting;
	size_t ciphertextLength, messageLength = strlen(message), result_len;
	char **c = (char**)calloc(sizeof(char *), 1);
	char **m_result = (char**)calloc(sizeof(char *), 1);
	int fail = 0;
	public_key* publicKey = (public_key*)calloc(sizeof(public_key), 1);
	private_key* privateKey = (private_key*)calloc(sizeof(private_key), 1);

	printf("\n_______________________Key generation_______________________\n\n");

	// Initialize public and private key
	mpz_init(publicKey->n);
	mpz_init(publicKey->e);
	mpz_init(privateKey->n);
	mpz_init(privateKey->e);
	mpz_init(privateKey->d);
	mpz_init(privateKey->p);
	mpz_init(privateKey->q);

	clock_t keygenTime = std::clock();
	rsaKeyGeneration(publicKey, privateKey);
	printTime(keygenTime);

	printf("\n_________________________Encription_________________________\n\n");

	rsaEncryption(publicKey, message, messageLength, c, &ciphertextLength);

	printf("\n_________________________Decription_________________________\n\n");

	rsaDecryption(privateKey, *c, ciphertextLength, m_result, &result_len);

	printf("\n________________________Final Result________________________\n\n");
	printf("expected:\n'%s' \ngot:\n'%s'\n", message, *m_result);

	free(publicKey);
	free(privateKey);
	free(*c);
	free(*m_result);

	if (fail == 0) {
		printf("\nTest PASSED\n");
	}
}

void printGPUProperties() {
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		cout << "  Number of multiprocesors: " << prop.multiProcessorCount << endl;
		cout << "  Total global memory: " << prop.totalGlobalMem << endl;
		cout << "  Shared memory per block: " << prop.sharedMemPerBlock << endl;
		cout << "  Number of registers per block per block: " << prop.regsPerBlock << endl;
		cout << "  Warp size: " << prop.warpSize << endl;
		cout << "  Max threads per block: " << prop.maxThreadsPerBlock << endl;
	}
	cout << endl;
}

void printSomeDebuggingStuff() {
	printf("CHAR_BIT je: %d\n", CHAR_BIT);
	printf("Velicina char je: %d\n", sizeof(char));
	printf("Velicina unsigned char je: %d\n", sizeof(unsigned char));
	printf("Velicina unsigned short je: %d\n", sizeof(unsigned short));
	printf("Velicina unsigned int je: %d\n", sizeof(unsigned int));
	printf("Velicina mp_limb_t je: %d\n", sizeof(mp_limb_t));
	printf("Velicina size_t je: %d\n", sizeof(size_t));
	printf("Velicina unsigned long int je: %d\n", sizeof(unsigned long int));
	printf("Velicina unsigned long long je: %d\n", sizeof(unsigned long long));
	printf("Velicina unsigned long long int je: %d\n", sizeof(unsigned long long int));
}

int main() {

	printGPUProperties();
	printSomeDebuggingStuff();

	testRsaSequentialMontgomery();

	return 0;
}