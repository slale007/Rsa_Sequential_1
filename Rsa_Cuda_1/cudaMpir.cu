#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "cudaMpir.h"
#include ""

// #define LONGINT int

using namespace std;
/*
__global__ void cuda_RightShiftsBlocks(LONGINT* result, LONGINT* inputNumber, int inputNumberLength, int shift) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < inputNumberLength) {
		int target = idx - shift;
		if (target >= 0 ) {
			result[target] = inputNumber[idx];
		}
	}
}

void RightShiftBlocks(mpz_t result, mpz_t inputNumber, int shift) {
	hipError_t cudaStatus;

	if (inputNumber->_mp_size <= shift) {
		result->_mp_size = 1;
		result->_mp_alloc = 1;
		result->_mp_d = (unsigned long long int *)malloc(sizeof(unsigned long long int));
		result->_mp_d[0] = 0;
		return;
	}

	result->_mp_size = inputNumber->_mp_size - shift;
	result->_mp_alloc = result->_mp_size;
	result->_mp_d = (unsigned long long int *)malloc(result->_mp_size * sizeof(unsigned long long int));
	LONGINT* dev_inputNumber;
	LONGINT* dev_result;

	cudaStatus = hipMalloc((void**)&dev_inputNumber, inputNumber->_mp_size * sizeof(unsigned long long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed0!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_result, result->_mp_size * sizeof(unsigned long long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed1!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_inputNumber, inputNumber->_mp_d, inputNumber->_mp_size * sizeof(unsigned long long int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed2!");
		goto Error;
	}

	int inputNumberLength = inputNumber->_mp_size;

	// cudaStatus = hipMemset(dev_result, 0, result->_mp_size * sizeof(unsigned long long int));

	// Launch a kernel on the GPU with one thread for each element.
	cuda_RightShiftsBlocks<<<1, inputNumberLength >>>(dev_result, dev_inputNumber, inputNumberLength, shift);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "0cuda_RightShiftsBlocks launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_RightShiftsBlocks!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result->_mp_d, dev_result, result->_mp_size * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_inputNumber);
	hipFree(dev_result);
}



__global__ void cuda_Multiplication(LONGINT* result, unsigned char* first, unsigned char* second, int lengthFirst, int lengthSecond) {

int idx = blockIdx.x * blockDim.x + threadIdx.x;

__shared__ unsigned char shared_First[2048];
__shared__ unsigned char shared_Second[2048];

for (int i = threadIdx.x; i < lengthFirst; i += blockDim.x) {
shared_First[i] = first[i];
}

for (int i = threadIdx.x; i < lengthSecond; i += blockDim.x) {
shared_Second[i] = second[i];
}

__syncthreads();

if (idx < lengthFirst) {
int m = 0;
int n = idx;
int tmp = 0;

while (n >= 0 && m < lengthSecond) {
tmp += shared_Second[m] * shared_First[n];
m++;
n--;
}

result[idx] = tmp;
}
else if (idx < lengthFirst + lengthSecond - 1) {
int n = lengthFirst - 1;
int m = idx - n;
int tmp = 0;

while (m < lengthSecond && n >= 0) {
tmp += shared_Second[m] * shared_First[n];
m++;
n--;
}

result[idx] = tmp;
}
}

__global__ void cuda_CarryUpdate(LONGINT* longResult, int* lengthLongResult) {

int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx == 0) {
int len = *lengthLongResult;
int carry = 0;
int tmp = 0;
int i;
for (i = 0; i < len; i++) {
tmp = longResult[i] + carry;
carry = tmp >> 8;
longResult[i] = tmp & 0xff;
}

if (carry != 0) {
longResult[i] = carry;
*lengthLongResult = i + 1;
}
}
}

// Is this tested?
//
////
void MultiplicationInCuda(mpz_t result, mpz_t first, mpz_t second) {
hipError_t cudaStatus;
int length1 = first->_mp_size * 8;
int length2 = second->_mp_size * 8;

mpz_init(result);
result->_mp_size = (length1 + length2);
result->_mp_alloc = result->_mp_size;
result->_mp_d = (unsigned long long int *)malloc(result->_mp_size * sizeof(unsigned long long int));

int *tmpRes = (int *)malloc((length1 + length2) * sizeof(int));

unsigned char* dev_first;
unsigned char* dev_second;
LONGINT* dev_result;

cudaStatus = hipMalloc((void**)&dev_result, (length1 + length2) * sizeof(LONGINT));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed-1!");
goto Error;
}

cudaStatus = hipMalloc((void**)&dev_first, length1 * sizeof(unsigned char));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed0!");
goto Error;
}

cudaStatus = hipMalloc((void**)&dev_second, length2 * sizeof(unsigned char));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed1!");
goto Error;
}

// Copy input vectors from host memory to GPU buffers.
cudaStatus = hipMemcpy(dev_first, first->_mp_d, length1 * sizeof(unsigned char), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy first failed!");
goto Error;
}

cudaStatus = hipMemcpy(dev_second, second->_mp_d, length2 * sizeof(unsigned char), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy second failed!");
goto Error;
}


int numberOfThreads = length1 + length2;

// Launch a kernel on the GPU with one thread for each element.
cuda_Multiplication <<< 1, numberOfThreads >>>(dev_result, dev_first, dev_second, length1, length2);


// Check for any errors launching the kernel
cudaStatus = hipGetLastError();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "Multiplication launch failed123: %s\n", hipGetErrorString(cudaStatus));
goto Error;
}

// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "2cudaDeviceSynchronize returned error code %d after launching cuda_RightShiftsBlocks!\n", cudaStatus);
goto Error;
}

// carry update on cuda

/*int *midLength;
int realMidLength = length1 + length2;


cudaStatus = hipMalloc((void**)&midLength, sizeof(int));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed0!");
goto Error;
}

cudaStatus = hipMemcpy(midLength, &realMidLength, sizeof(int), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy second failed!");
goto Error;
}

cuda_CarryUpdate <<<1, 1 >>> (dev_result, midLength);

// Check for any errors launching the kernel
cudaStatus = hipGetLastError();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "Multiplication launch failed1234: %s\n", hipGetErrorString(cudaStatus));
goto Error;
}

// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "23cudaDeviceSynchronize returned error code %d after launching cuda_RightShiftsBlocks!\n", cudaStatus);
goto Error;
}

// END carry update on cuda 




// Copy output vector from GPU buffer to host memory.
cudaStatus = hipMemcpy(tmpRes, dev_result, (length1 + length2) * sizeof(LONGINT), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
	fprintf(stderr, "2cudaMemcpy failed!\n");
	goto Error;
}

int len = length1 + length2 - 1;
unsigned long long int carry = 0;
unsigned long long int tmp = 0;
int i;
for (i = 0; i < len; i++) {
	tmp = tmpRes[i] + carry;
	carry = tmp >> 8;
	tmpRes[i] = tmp & 0xff;
}

if (carry != 0) {
	tmpRes[i] = carry;
	len++;
}

result->_mp_d[0] = tmpRes[0];
result->_mp_d[0] |= tmpRes[1] << 8;
result->_mp_d[0] |= (unsigned long long int)tmpRes[2] << 2 * 8;
result->_mp_d[0] |= (unsigned long long int)tmpRes[3] << 3 * 8;
result->_mp_d[0] |= (unsigned long long int)tmpRes[4] << 4 * 8;
result->_mp_d[0] |= (unsigned long long int)tmpRes[5] << 5 * 8;
result->_mp_d[0] |= (unsigned long long int)tmpRes[6] << 6 * 8;
result->_mp_d[0] |= (unsigned long long int)tmpRes[7] << 7 * 8;

for (int k = 1; k < (len + 1) / 8; k++) {
	result->_mp_d[k] = 0;
	result->_mp_d[k] |= tmpRes[8 * k];
	result->_mp_d[k] |= tmpRes[8 * k + 1] << 8;
	result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 2] << 2 * 8;
	result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 3] << 3 * 8;
	result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 4] << 4 * 8;
	result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 5] << 5 * 8;
	result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 6] << 6 * 8;
	result->_mp_d[k] |= (unsigned long long int)tmpRes[8 * k + 7] << 7 * 8;
}

result->_mp_size = (length1 + length2) / 8;
result->_mp_alloc = (length1 + length2) / 8;

Error:
hipFree(dev_first);
hipFree(dev_second);
hipFree(dev_result);
}



*/