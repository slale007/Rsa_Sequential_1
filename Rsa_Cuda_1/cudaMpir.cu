#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "cudaMpir.h"
#include ""

#define LONGINT unsigned long long int


__global__ void cuda_RightShiftsBlocks(LONGINT* result, LONGINT* inputNumber, int inputNumberLength, int shift) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < inputNumberLength) {
		int target = idx - shift;
		if (target >= 0 ) {
			result[target] = inputNumber[idx];
		}
	}
}

void RightShiftBlocks(mpz_t result, mpz_t inputNumber, int shift) {
	hipError_t cudaStatus;

	if (inputNumber->_mp_size <= shift) {
		result->_mp_size = 1;
		result->_mp_alloc = 1;
		result->_mp_d = (unsigned long long int *)malloc(sizeof(unsigned long long int));
		result->_mp_d[0] = 0;
		return;
	}

	result->_mp_size = inputNumber->_mp_size - shift;
	result->_mp_alloc = result->_mp_size;
	result->_mp_d = (unsigned long long int *)malloc(result->_mp_size * sizeof(unsigned long long int));
	LONGINT* dev_inputNumber;
	LONGINT* dev_result;

	cudaStatus = hipMalloc((void**)&dev_inputNumber, inputNumber->_mp_size * sizeof(unsigned long long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed0!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_result, result->_mp_size * sizeof(unsigned long long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed1!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_inputNumber, inputNumber->_mp_d, inputNumber->_mp_size * sizeof(unsigned long long int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed2!");
		goto Error;
	}

	int inputNumberLength = inputNumber->_mp_size;

	// cudaStatus = hipMemset(dev_result, 0, result->_mp_size * sizeof(unsigned long long int));

	// Launch a kernel on the GPU with one thread for each element.
	cuda_RightShiftsBlocks<<<1, 64 >>>(dev_result, dev_inputNumber, inputNumberLength, shift);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda_RightShiftsBlocks launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_RightShiftsBlocks!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result->_mp_d, dev_result, result->_mp_size * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_inputNumber);
	hipFree(dev_result);
}

void RightShift2(mpz_t result, mpz_t inputNumber, int shift) {

	RightShiftBlocks(result, inputNumber, shift);
}